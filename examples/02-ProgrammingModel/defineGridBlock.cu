#include<hip/hip_runtime.h>
#include<stdio.h>
#include"../common/common.h"

int main(){
    int nElem = 1024;

    dim3 block (1024);
    dim3 grid ((nElem+block.x-1)/block.x);
    printf("grid.x: %d, block.x:%d\n", grid.x, block.x);

    block.x = 512;
    grid.x = (nElem+block.x-1)/block.x;
    printf("grid.x: %d, block.x:%d\n", grid.x, block.x);

    block.x = 256;
    grid.x = (nElem+block.x-1)/block.x;
    printf("grid.x: %d, block.x:%d\n", grid.x, block.x);

    block.x = 128;
    grid.x = (nElem+block.x-1)/block.x;
    printf("grid.x: %d, block.x:%d\n", grid.x, block.x);

    CHECK(hipDeviceReset());
    return 0;
}
